
// program pokazujacy dodawanie wektorow na GPU
// kompilacja:
// nvcc vecAdd.cu -o vecAdd
// w programie ZAKLADAMY, ze wektor na dlugosc 10

#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 10

__global__ void vecAdd( float*A, float*B, float*C ){
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main(){

	float *hostA = (float*) malloc(SIZE*sizeof(float));
	float *hostB = (float*) malloc(SIZE*sizeof(float));
	float *hostC = (float*) malloc(SIZE*sizeof(float));
	float *devA,*devB,*devC;

	// ustawiamy wartosci wektor na hoscie na 1
	int i;
	for(i=0;i<SIZE;i++){
		hostA[i] = 1;
		hostB[i] = 1;
	}

	// robimy alokacje pamieci na GPU
	// http://pl.wikipedia.org/wiki/Alokacja_zasob%C3%B3w
	hipMalloc( (void**) &devA, sizeof(float)*SIZE);
	hipMalloc( (void**) &devB, sizeof(float)*SIZE);
	hipMalloc( (void**) &devC, sizeof(float)*SIZE);

	// przesylamy dane z wektorow host do dev
	// (czyli z hostA do devA, itd)
	hipMemcpy(devA,hostA,SIZE*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(devB,hostB,SIZE*sizeof(float),hipMemcpyHostToDevice);

	// dane sa przeslane do karty. Mamy hostA na CPU i devA na GPU
	// z CPU widac tylko wskaznik devA - ktory ma sens tylko na GPU
	// odwolania z CPU do danych wskazywanych przez devA moga zabic

	// odpalamy obliczenia na GPU, coby wykonac sumowanie wektorow
	vecAdd<<<1,SIZE>>>(devA,devB,devC);

	// po obliczeniach, nalezy sume wektorow WYSLAC z powrotem do CPU
	// czyli do wektora hostC
	hipMemcpy(hostC,devC,SIZE*sizeof(float),hipMemcpyDeviceToHost);

	// teraz trzeba wypisac, czy sie zsumowalo. Bo inaczej mozna tylko gdybac.
	// suma wektorow rownych 1 powinna wynosic 2 w kazdym elemencie.

	printf("SUMA hostC = ");
	for(i=0;i<SIZE;i++)	printf(" %1.f",hostC[i]);
	printf("\n");

	// zwalnianie pamieci wczesniej zaalokowanej
	free(hostA);free(hostB);free(hostC);
	hipFree(devA);hipFree(devB);hipFree(devC);
}


