#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctype.h>

#define TILE_SIZE 2
#define MAX_SIZE 2500


__global__ void MatrixMulShared(int *M, int *N, int *P, int width) {

  __shared__ int sharedM[TILE_SIZE][TILE_SIZE];
  __shared__ int sharedN[TILE_SIZE][TILE_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_SIZE + ty;
  int col = bx * TILE_SIZE + tx;

  int sum = 0;

  for(int m = 0; m < width/TILE_SIZE; m++) {
    sharedM[ty][tx] = M[row*width + (m*TILE_SIZE + tx)];
    sharedN[ty][tx] = N[(m*TILE_SIZE + ty)*width + col];

    __syncthreads();

    for(int k = 0; k < TILE_SIZE; k++) {
      sum += sharedM[m+ty][k] * sharedN[k][m+tx]; 
    }
    __syncthreads();
  }
  P[row*width+col] = sum; 
}


void printMat(int *a, int width) {
    int i,j;
    for(i=0;i<width;i++) 
        for(j=0;j<width;j++)
            printf("%d%c",a[i*width+j],(j == (width-1))?'\n':'\t');

    printf("\n");
}


int* generatorMatrixa(int width) {
    int zarodek;
    zarodek= time(NULL);
    srand(zarodek);   // za zarodek wstawiamy pobrany czas w sekundach

    int *a = (int*)calloc(width*width, sizeof(int));
    int i,j;
    for(i=0;i<width;i++)
        for(j=0;j<width;j++)
            a[i*width+j] = rand()%10;
    return a;
}


int *matMul(int *hostA, int *hostB, int width) {

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    int SIZE = width*width;
    int *hostC = (int*)calloc(SIZE,sizeof(int));

    int *devA, *devB, *devC;
    hipMalloc((void**) &devA, SIZE*sizeof(int));
    hipMalloc((void**) &devB, SIZE*sizeof(int));
    hipMalloc((void**) &devC, SIZE*sizeof(int));

       dim3 gridDim(width/TILE_SIZE, width/TILE_SIZE);
       dim3 blockDim(TILE_SIZE,TILE_SIZE);
    hipMemcpy(devA, hostA, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devC, hostC, SIZE*sizeof(int), hipMemcpyHostToDevice);


       hipEventRecord(start, 0); //rozpoczecie pomiaru czasu
       MatrixMulShared<<<gridDim,blockDim>>>(devA, devB, devC, width); //AKCJA!
       hipEventRecord(stop, 0); // koniec pomiaru
       hipEventSynchronize(stop); // synchro

    hipMemcpy(hostC, devC, SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&time, start, stop);
    printf ("Czas trwania: %f ms\n", time);

    hipFree(devA);hipFree(devB);hipFree(devC);
    return hostC;
}

void oblicz(int Rozmiar){

    int rozmiar = Rozmiar;
    int *A = generatorMatrixa(rozmiar);
    int *B = generatorMatrixa(rozmiar);

    //inicjalizacja i jednoczesne obliczenia macierzy.
    printf("Dla macierzy rozmiarów  =  %i ", rozmiar);
    int *C = matMul(A,B,rozmiar);
 
    //printf("\n");
    //printMat(C,m);
   
    free(A);
    free(B);
    free(C);
}


int main() {
    
    //seryjka 
    int i;
    for ( i = 0; i < MAX_SIZE; ++i) if (i%100 == 0) {
      oblicz(i);
    }
    
   
    return 0;

}