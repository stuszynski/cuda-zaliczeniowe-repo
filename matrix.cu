#include "hip/hip_runtime.h"
#include <stdio.>
#include <hip/hip_runtime.h>

/* naive CUDA version */
__global__ void naiveMul( float* P, float* M, float* N, int width ) {
int tx = threadIdx.x; // current thread position
int ty = threadIdx.y; // in 2D block of threads
float sum = 0.0; // element computed by the thread

for (k = 0; k < width; k++)
	sum += M[ty*width+k] * N[k*width+tx];
P[ty*width+tx] = sum;
// element [tx,ty] is now calculated
}
...

int main(int argc, char const *argv[])
{
	dim3 blockDim( width, width );
	dim3 gridDim(1,1);
	naviveMul <<< gridDim, blockDim >>> ( P, M, N, width );
// <<< SIZE, 1 >>>
	return 0;
}



